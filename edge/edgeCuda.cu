#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>

// unsigned char *edge = matrice de sortie, g = matrice d'entree
__global__ void edge(unsigned char *edge, unsigned char *g, std::size_t cols, std::size_t rows)
{
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < cols && j < rows && i > 1 && j > 1)
  {

    // un pixel est calculé à partir de ses 8 voisins
    auto res = -1 * g[((j - 1) * cols + i - 1)] + -1 * g[((j - 0) * cols + i - 1)] + -1 * g[((j + 1) * cols + i - 1)] +
               -1 * g[((j - 1) * cols + i - 0)] + 8 * g[((j - 0) * cols + i + 0)] + -1 * g[((j + 1) * cols + i + 0)] +
               -1 * g[((j - 1) * cols + i + 1)] + -1 * g[((j - 0) * cols + i + 1)] + -1 * g[((j + 1) * cols + i + 1)];

    // on normalise le pixel (pas d'overshoot / undershoot )
    res = res > 255 ? res = 255 : res;
    res = res < 0 ? res = 0 : res;

    edge[j * cols + i] = res;
  }
}

int main(int argc, char **argv)
{
  
  // lecture du fichier avec openCV, on récupère une image en niveau de gris
  cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  // version applatie de l'image de sortie
  std::vector<unsigned char> g(rows * cols);

  // on crée une image avec opencv à partir du vecteur (CV_8UC1 : les pixels sont en 8bit et monocouleur)
  cv::Mat m_out(rows, cols, CV_8UC1, g.data());

  hipEvent_t start, stop;

  // pour le monitoring du temps mis par le kernel
  hipEventCreate(&start);
  hipEventCreate(&stop);

  unsigned char *g_d;
  unsigned char *edge_d;

  // allocation de la mémoire sur le GPU
  hipMalloc(&g_d, rows * cols);
  hipMalloc(&edge_d, rows * cols);

  // copie de l'image de l'hôte vers le GPU
  hipMemcpy(g_d, rgb, rows * cols, hipMemcpyHostToDevice);


  dim3 grid(32, 32);
  dim3 block((cols - 1) / grid.x + 1, (rows - 1) / grid.y + 1);

  hipEventRecord(start);

  // lancement du kernel
  edge<<<block, grid>>>(edge_d, g_d, cols, rows);

  hipEventRecord(stop);

  // copie de l'image du GPU vers l'hôte
  hipMemcpy(g.data(), edge_d, rows * cols, hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  cv::imwrite("outcuda.jpg", m_out);

  std::cout << ms << std::endl;

  hipFree(g_d);

  // gestion d'une éventuelle erreur
  if (hipGetLastError() != hipSuccess)
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
  return 0;
}

#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>

__global__ void line(unsigned char *out, unsigned char *in, std::size_t cols, std::size_t rows)
{
    int i = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
    int j = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

    int shared_i = threadIdx.x;
    int shared_j = threadIdx.y;

    int w = blockDim.x;
    int h = blockDim.y;

    /* shared_g est la mémoire partagée et est initialisée à partir d'un paramètre donné au kernel avec le mot clé extern.
      Même si cela est plus souple que de hardcoder la taille, on ne peut pas créer de tableaux 2D.
    */
    extern __shared__ unsigned char shared_g[];

    // on remplie la mémoire partagée avec l'image donnée en entrée du kernel
    if (i < cols && j < rows)
    {
        shared_g[shared_j * w + shared_i] = in[j * cols + i];
    }

    // on s'assure que toute l'image a été copiée dans la mémoire partagée avant de la lire
    __syncthreads();

    if (i < cols - 1 && j < rows - 1 && shared_i > 0 && shared_i < (w - 1) && shared_j > 0 && shared_j < (h - 1))
    {
        // un pixel est calculé à partir de ses 8 voisins
        auto res = -1 * in[((j - 1) * cols + i - 1)] + -1 * in[((j - 0) * cols + i - 1)] + 2 * in[((j + 1) * cols + i - 1)] +
                   -1 * in[((j - 1) * cols + i - 0)] + 2 * in[((j - 0) * cols + i + 0)] + -1 * in[((j + 1) * cols + i + 0)] +
                   2 * in[((j - 1) * cols + i + 1)] + -1 * in[((j - 0) * cols + i + 1)] + -1 * in[((j + 1) * cols + i + 1)];

        res = min(255, max(0, res));
        out[j * cols + i] = res;
    }
}

int main(int argc, char **argv)
{
    // lecture du fichier avec openCV, on récupère une image en niveau de gris
    cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    // version applatie de l'image de sortie
    std::vector<unsigned char> g(rows * cols);

    // on crée une image avec opencv à partir du vecteur (CV_8UC1 : les pixels sont en 8bit et monocouleur)
    cv::Mat m_out(rows, cols, CV_8UC1, g.data());

    unsigned char *g_d;
    unsigned char *line_d;

    // allocation de la mémoire sur le GPU
    hipMalloc(&g_d, rows * cols);
    hipMalloc(&line_d, rows * cols);

    // copie de l'image de l'hôte vers le GPU
    hipMemcpy(g_d, m_in.data, rows * cols, hipMemcpyHostToDevice);
    hipEvent_t start, stop;

    // pour le monitoring du temps mis par le kernel
    hipEventCreate(&start);
    hipEventCreate(&stop);
    dim3 block(32, 4);

    dim3 grid((cols - 1) / (block.x - 2) + 1, (rows - 1) / (block.y - 2) + 1);

    hipEventRecord(start);

    // lancement du kernel
    line<<<grid, block, block.x * block.y>>>(line_d, g_d, cols, rows);

    hipEventRecord(stop);

    hipMemcpy(g.data(), line_d, rows * cols, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    cv::imwrite("outshared.jpg", m_out);

    std::cout << ms << std::endl;
    hipFree(g_d);
    hipFree(line_d);

    // gestion d'une éventuelle erreur
    if (hipGetLastError() != hipSuccess)
        std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

    return 0;
}

#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <chrono>

__constant__ double kernel[49] = {0, 0, 0, 5, 0, 0, 0,
                                  0, 5, 18, 32, 18, 5, 0,
                                  0, 18, 64, 100, 64, 18, 0,
                                  5, 32, 100, 100, 100, 32, 5,
                                  0, 18, 64, 100, 64, 18, 0,
                                  0, 5, 18, 32, 18, 5, 0,
                                  0, 0, 0, 5, 0, 0, 0};

// unsigned char *gaussian = matrice de sortie, g = matrice d'entree
__global__ void gaussian(unsigned char *gaussian, unsigned char *g, std::size_t cols, std::size_t rows)
{
  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  auto y = blockIdx.y * blockDim.y + threadIdx.y;

  int tailleKernel = 7;

  if (x < cols && y < rows && x > 0 && y > 0)
  {

    double res = 0;
    double sommeKernel = 0;
    for (int j = -tailleKernel / 2; j <= tailleKernel / 2; ++j)
    {
      for (int i = -tailleKernel / 2; i <= tailleKernel / 2; ++i)
      {
        // coordonnées du pixel qui se fait filtrer
        int yy = y + j;
        int xx = x + i;
        if (yy < 0 || yy >= rows || xx < 0 || xx >= cols)
          continue;
        double weight = kernel[(j + tailleKernel / 2) * tailleKernel + i + tailleKernel / 2];
        res += weight * g[yy * cols + xx];
        sommeKernel += weight;
      }
    }

    gaussian[y * cols + x] = res / sommeKernel;
  }
}

int main(int argc, char **argv)
{
  cv::Mat m_in = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);

  auto rows = m_in.rows;
  auto cols = m_in.cols;
  int nb_pixels = rows * cols;

  std::vector<unsigned char> g(rows * cols);

  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  unsigned char *g_d;
  unsigned char *gaussian_d;
  unsigned char *pixels;
  unsigned char *res;

  hipHostMalloc(&pixels, nb_pixels);
  hipHostMalloc(&res, nb_pixels);

  pixels = m_in.data;

  hipMalloc(&g_d, nb_pixels + cols);
  hipMalloc(&gaussian_d, nb_pixels + cols);

  hipStream_t streams[2];

  hipStreamCreate(&streams[0]);
  hipStreamCreate(&streams[1]);

  hipMemcpyAsync(g_d, pixels, nb_pixels / 2 + cols, hipMemcpyHostToDevice, streams[0]);
  hipMemcpyAsync(g_d + nb_pixels / 2, pixels + (nb_pixels / 2) - cols, nb_pixels / 2 + cols, hipMemcpyHostToDevice, streams[0]);

  dim3 t(32, 32);
  dim3 b((cols - 1) / t.x + 1, (rows - 1) / t.y + 1);

  hipEventRecord(start, streams[0]);

  gaussian<<<b, t, 0, streams[0]>>>(gaussian_d, g_d, cols, (rows / 2) + 1);
  gaussian<<<b, t, 0, streams[1]>>>(gaussian_d + nb_pixels / 2, g_d + nb_pixels / 2 - cols, cols, (rows / 2) - 1);

  hipEventRecord(stop, streams[0]);

  hipMemcpyAsync(res, gaussian_d, nb_pixels / 2, hipMemcpyDeviceToHost, streams[1]);
  hipMemcpyAsync(res + nb_pixels / 2, gaussian_d + nb_pixels / 2 + cols, nb_pixels / 2, hipMemcpyDeviceToHost, streams[1]);

  hipDeviceSynchronize();

  hipEventSynchronize(stop);

  hipStreamDestroy(streams[0]);
  hipStreamDestroy(streams[1]);

  cv::Mat m_out(rows, cols, CV_8UC1, res);

  float ms = 0;
  hipEventElapsedTime(&ms, start, stop);
  cv::imwrite("outStream.jpg", m_out);

  std::cout << ms << std::endl;

  hipFree(g_d);
  hipFree(gaussian_d);
  hipHostFree(pixels);
  hipHostFree(res);

  if (strcmp(hipGetErrorString(hipGetLastError()), "no error") != 0)
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;

  return 0;
}